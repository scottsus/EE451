
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

__global__ void matrixMult(int *A, int *B, int *C)
{
    printf("Sequential Matrix Mult\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
            {
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}

void log(int *A)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", A[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char **argv)
{
    const int bytes = N * N * sizeof(int);

    int deviceId = 0;
    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties(&prop, deviceId));
    printf("Device: %s\n", prop.name);
    checkCuda(hipSetDevice(deviceId));

    int *A, *B, *C, *d_A, *d_B, *d_C;
    checkCuda(hipHostMalloc((void **)&A, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc((void **)&B, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc((void **)&C, bytes, hipHostMallocDefault));
    checkCuda(hipMalloc((void **)&d_A, bytes));
    checkCuda(hipMalloc((void **)&d_B, bytes));
    checkCuda(hipMalloc((void **)&d_C, bytes));
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = i;
            B[i * N + j] = j;
            C[i * N + j] = 0;
        }
    }

    float ms;
    hipEvent_t startEvent, stopEvent, dummyEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    checkCuda(hipEventCreate(&dummyEvent));

    checkCuda(hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(startEvent, 0));
    checkCuda(hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_C, C, bytes, hipMemcpyHostToDevice));

    dim3 dimGrid(64, 64);
    dim3 dimBlock(16, 16);

    matrixMult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    checkCuda(hipMemcpy(A, d_A, bytes, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost));
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

    printf("Serial time (ms): %f\n", ms);
    printf("C[451][451]: %d\n", C[451 * N + 451]);

    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
    checkCuda(hipEventDestroy(dummyEvent));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    printf("Program exiting\n");
    return 0;
}
