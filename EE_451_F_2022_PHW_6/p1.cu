
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

__global__ void matrixMult(int *A, int *B, int *C, int offset, int streamSize)
{
    for (int i = offset; i < offset + streamSize; i++)
    {
        for (int j = 0; j < N; j++)
        {
            for (int k = 0; k < N; k++)
            {
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}

void log(int *A)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", A[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char **argv)
{
    const int nStreams = 4;
    const int streamSize = N * N / nStreams;
    const int streamBytes = streamSize * sizeof(int);
    const int bytes = N * N * sizeof(int);

    int deviceId = 0;
    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties(&prop, deviceId));
    printf("Device: %s\n", prop.name);
    checkCuda(hipSetDevice(deviceId));

    int *A, *B, *C, *d_A, *d_B, *d_C;
    checkCuda(hipHostMalloc((void **)&A, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc((void **)&B, bytes, hipHostMallocDefault));
    checkCuda(hipHostMalloc((void **)&C, bytes, hipHostMallocDefault));
    checkCuda(hipMalloc((void **)&d_A, bytes));
    checkCuda(hipMalloc((void **)&d_B, bytes));
    checkCuda(hipMalloc((void **)&d_C, bytes));
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = i;
            B[i * N + j] = j;
            C[i * N + j] = 0;
        }
    }

    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++)
        checkCuda(hipStreamCreate(&stream[i]));

    float ms;
    hipEvent_t startEvent, stopEvent, dummyEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    checkCuda(hipEventCreate(&dummyEvent));

    dim3 dimGrid(64 / nStreams, 64);
    dim3 dimBlock(16, 16);

    checkCuda(hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(startEvent, 0));
    for (int i = 0; i < nStreams; i++)
    {
        int offset = i * streamSize;
        checkCuda(hipMemcpyAsync(&d_A[offset], &A[offset], streamBytes, hipMemcpyHostToDevice, stream[i]));
        checkCuda(hipMemcpyAsync(&d_C[offset], &C[offset], streamBytes, hipMemcpyHostToDevice, stream[i]));

        matrixMult<<<dimGrid, dimBlock, 0, stream[i]>>>(d_A, d_B, d_C, offset, streamSize);

        checkCuda(hipMemcpyAsync(&A[offset], &d_A[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
        checkCuda(hipMemcpyAsync(&C[offset], &d_C[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]));
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
    checkCuda(hipMemcpy(B, d_B, bytes, hipMemcpyDeviceToHost));

    printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
    printf("C[451][451]: %d\n", C[451 * N + 451]);

    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
    checkCuda(hipEventDestroy(dummyEvent));
    for (int i = 0; i < nStreams; i++)
        checkCuda(hipStreamDestroy(stream[i]));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    printf("Program exiting\n");
    return 0;
}
